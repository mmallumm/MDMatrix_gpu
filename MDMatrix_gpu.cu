#include "hip/hip_runtime.h"
#include "MDMatrix_gpu.h"
#include "..\..\Core\MDMatrix.h"
#include <math.h>
#include <iostream>

#define MAXGRIDDIM 65535
#define imin(a,b) (a<b?a:b)
#define imax(a, b) (a>b?a:b)

namespace RDK {

    //--------------------------------kernels' block--------------------

    //FillMatrixValue----------
    template<class T>
    __global__ void _FillMatrixValue(T* data, T defValue, int rows, int cols) {
        int tid = threadIdx.x;
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        int step = gridDim.x * blockDim.x;
        int size = rows * cols;

        //in case of kernels' number > matrix' size
        if(step >= size) {
            if(id < size) {
                data[id] = defValue;
            }
        } else {
            for(int c = id; c < size; c += step) {
                if(c < size) {
                    data[c] = defValue;
                }
            }
        }
    }

    //FillMatrixFrom----------
    template<class T>
    __global__ void _FillMatrixFrom(T* dest_data, T* src_data, int rows, int cols) {
        int tid = threadIdx.x;
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        int step = gridDim.x * blockDim.x;
        int size = rows * cols;

        //in case of kernels' number > matrix' size
        if(step >= size) {
            if(id < size) {
                dest_data[id] = src_data[id];
            }
        } else {
            for(int c = id; c < size; c += step) {
                if(c < size) {
                    dest_data[c] = src_data[c];
                }
            }
        }
    }

    //FillMatrixValue_2dim-----
    template<class T>
    __global__ void _FillMatrixValue_2dim(T* dest_data, T defValue, int rows, int cols, int start_row, int start_col) {
        int tid = threadIdx.x + threadIdx.y * blockDim.x;
        int bid = blockIdx.x + blockIdx.y * gridDim.x;
        int nthreads = blockDim.x * blockDim.y; //number of threads per block
        int id = tid + nthreads * bid; //absolute id
        int step_x = blockDim.x * gridDim.x;;
        int step_y = blockDim.y * gridDim.y;
        int size = cols * rows;

        int c_row = blockIdx.y * blockDim.y + threadIdx.y; //current row
        int c_col = blockIdx.x * blockDim.x + threadIdx.x; //current col

        if(step_x * step_y >= size) {
            if(id < size && c_row >= start_row && c_col >= start_col) {
                dest_data[c_row * cols + c_col] = defValue;
            }
        } else {
            for(int i = c_row; i < rows; i += step_y) {
                for(int j = c_col; j < cols; j += step_x) {
                    if(i >= start_row && j >= start_col && i*j < size)
                        dest_data[i * cols + j] = defValue;
                }
            }
        }
    }

    //_AddMatrix
    остановился тут
    template<class T>
    __global__ void _AddMatrix(T* dest_data, T* sec_data, size_t size)
    {
        size_t thId = threadIdx.x;
        size_t id = blockDim.x * blockIdx.x + threadIdx.x;
        size_t step = gridDim.x * blockDim.x;

        if(step >= size) {
            if(id < size) {
                dest_data[id] += sec_data[id];
            }
        } else {
            for(int c = id; c < size; c += step) {
                if(c < size) {
                    dest_data[c] += sec_data[c];
                }
            }
        }
    }

    //_SubMatrix
    template<class T>
    __global__ void _SubMatrix(T* dest_data, T* sec_data, size_t size)
    {
        size_t thId = threadIdx.x;
        size_t id = blockDim.x * blockIdx.x + threadIdx.x;
        size_t step = gridDim.x * blockDim.x;

        if(step >= size) {
            if(id < size) {
                dest_data[id] -= sec_data[id];
            }
        } else {
            for(int c = id; c < size; c += step) {
                if(c < size) {
                    dest_data[c] -= sec_data[c];
                }
            }
        }
    }

    template<class T>
    __global__ void _SetElement(T* data, int stride, int i, int j, T value)
    {
        data[i * stride + j] = value;
    }

    template<class T>
    __device__ T _GetElement(_DeviceMatrix<T>& A, int i, int j)
    {
        return A.Data[i * A.Stride + j];
    }

    template<class T>
    __device__ MDMatrix_gpu<T> _GetSubMatrix(_DeviceMatrix<T> A, int i, int j)
    {
        _DeviceMatrix<T> ASub;
        ASub.Cols = BLOCKSIZE;
        ASub.Rows = BLOCKSIZE;
        ASub.Stride = A.Stride;
        ASub.Data = &A.Data[A.Stride * BLOCKSIZE * i + BLOCKSIZE * j];
        return ASub;
    }

    template<class T>
    __global__ void _BadMatMul(const _DeviceMatrix<T> A, const _DeviceMatrix<T> B, _DeviceMatrix<T> C)
    {
        int ROW = blockIdx.y * blockDim.y + threadIdx.y;
        int COL = blockIdx.x * blockDim.x + threadIdx.x;
        T tmpSum = 0;

        if (ROW < A.Rows && COL < B.Cols) {
            for (int i = 0; i < A.Cols; i++) {  //A.Cols==B.Rows
                tmpSum += A.Data[ROW * A.Cols + i] * B.Data[i * B.Cols + COL];
            }
            C.Data[ROW * C.Cols + COL] = tmpSum;
        }
        __syncthreads();
    }



    //-----------------------------MDMatrix_gpu block--------------------
    //constructors
    //MDMatrix_gpu(int rows, int cols, T defvalue)
    template<class T>
    MDMatrix_gpu<T>::MDMatrix_gpu(int rows, int cols, T defvalue)
        : Data(0), Rows(0), Cols(0), Capacity(0)
    {
        GetDeviceProps();
        Resize(rows, cols);
        unsigned int N = rows * cols;
        unsigned int threadsPerBlock = _DeviceProps[0].maxThreadsDim[0];
        unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        _FillMatrixValue << <blocksPerGrid, threadsPerBlock >> > (Data, defvalue, rows, cols);
    };

    //MDMatrix_gpu(const MDMatrix_gpu<T>& other)
    template<class T>
    MDMatrix_gpu<T>::MDMatrix_gpu(const MDMatrix_gpu<T>& other)
        : Data(0), Rows(0), Cols(0), Capacity(0)
    {
        GetDeviceProps();
        *this = other;
    };

    //MDMatrix_gpu(const MDMatrix_gpu<T>&& other)
    template<class T>
    MDMatrix_gpu<T>::MDMatrix_gpu(const MDMatrix_gpu<T>&& other)
        : Data(0), Rows(0), Cols(0), Capacity(0)
    {
        GetDeviceProps();
        *this = other;
    };

    template<class T>
    MDMatrix_gpu<T>::MDMatrix_gpu(const _DeviceMatrix<T>& other)
        : Data(other.Data), Rows(other.Rows), Cols(other.Cols), Capacity(other.Capacity)
    {
        GetDeviceProps();
    }

    //methods
    //Resize(int rows, int cols, T defvalue);
    template <class T>
    void MDMatrix_gpu<T>::Resize(int rows, int cols, T defvalue) {
        if (rows == Rows && cols == Cols)
            return;

        if (rows < 0 || cols < 0)
            return;

        if (rows == 0 || cols == 0)
        {
            Rows = rows;
            Cols = cols;
            return;
        }

        T* new_data(0);
        int new_capacity(0);

        if (rows && cols && rows * cols > Capacity)
        {
            if (cols > Cols || rows > Rows + RDK_MMATRIX_RESIZE_ROW_LINEAR_CONSTANT)
                new_capacity = int(rows * cols * RDK_MMATRIX_RESIZE_AMORTIZED_CONSTANT);
            else
                new_capacity = (rows + RDK_MMATRIX_RESIZE_ROW_LINEAR_CONSTANT) * cols;

            unsigned int N = rows * cols;
            unsigned int threadsPerBlock = _DeviceProps[0].maxThreadsDim[0];
            unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
            dim3 grid = dim3(blocksPerGrid, 1, 1);
            dim3 block = dim3(threadsPerBlock, 1, 1);
            hipMalloc((void**)&new_data, sizeof(T) * new_capacity);

            if (!Data) // �������� ������ � ������� �� ����
            {
                if (!defvalue)
                    hipMemset(new_data, 0, rows * cols * sizeof(T));
                else
                {
                    _FillMatrixValue << <grid, block >> > (new_data, defvalue, rows, cols);
                }
            }
            else {
                if (new_data) // ������� �����������, �������� ����� ������, ��������� ������, ������� ��� ��������)
                {
                    int c_rows = (Rows < rows) ? Rows : rows;
                    int c_cols = (Cols < cols) ? Cols : cols;

                    //�������� �����������: ����� �� ��������� ��� ������� ���������� ��-���������
                    _FillMatrixValue << <grid, block >> > (new_data, defvalue, rows, cols);
                    _FillMatrixFrom << <grid, block >> > (new_data, Data, c_rows, c_cols);
                }
            }

        }
        else
            if (cols < Cols) // ������� ����������� �� ����� �������, ��������� ������ � ������ ������
            {
                T* old_pos = Data + 1 * Cols;
                T* new_pos = Data + cols;
                T* buff;
                hipMalloc((void**)&buff, sizeof(T) * cols);

                for (int i = 1; i < rows; i++)
                {
                    hipMemcpy(buff, old_pos, sizeof(T) * cols, hipMemcpyDeviceToDevice);
                    hipMemcpy(new_pos, buff, sizeof(T) * cols, hipMemcpyDeviceToDevice);
                    new_pos += cols;
                    old_pos += Cols;
                }

                // �������� �����������
                dim3 grid = dim3(255, 255, 1);
                dim3 block = dim3(32, 32, 1);
                _FillMatrixValue_2dim << <grid, block >> > (Data, defvalue, rows, cols, Rows, 0);
                _FillMatrixValue_2dim << <grid, block >> > (Data, defvalue, rows, cols, 0, Cols);

                hipFree(buff);
            }

            else // ������� ����������� ��� ����������, ��������� ������ � ������ ������
            {
                int c_rows = (Rows < rows) ? Rows : rows;
                T* old_pos = Data + (c_rows - 1) * Cols;
                T* new_pos = Data + (c_rows - 1) * cols;
                T* buff;
                hipMalloc((void**)&buff, sizeof(T) * Cols);

                for (int i = c_rows; i > 0; i--)
                {
                    hipMemcpy(buff, old_pos, sizeof(T) * Cols, hipMemcpyDeviceToDevice);
                    hipMemcpy(new_pos, buff, sizeof(T) * Cols, hipMemcpyDeviceToDevice);
                    new_pos -= cols;
                    old_pos -= Cols;
                }
                dim3 grid = dim3(255, 255, 1);
                dim3 block = dim3(32, 32, 1);
                _FillMatrixValue_2dim << <grid, block >> > (Data, defvalue, rows, cols, Rows, 0);
                _FillMatrixValue_2dim << <grid, block >> > (Data, defvalue, rows, cols, 0, Cols);

                hipFree(buff);
            }


        if (new_data)
        {
            hipFree(Data);
            Data = new_data;
            Capacity = new_capacity;
        }
        Rows = rows;
        Cols = cols;
    }

    template<class T>
    void MDMatrix_gpu<T>::SetElement(int i, int j, T value)
    {
        if (i >= Rows, j >= Cols)
            return;
        _SetElement << <1, 1 >> > (Data, Cols, i, j, value);
    }

    template<class T>
    int MDMatrix_gpu<T>::GetCols() const
    {
        return Cols;
    }

    template<class T>
    int MDMatrix_gpu<T>::GetRows() const
    {
        return Rows;
    }

    template<class T>
    T* MDMatrix_gpu<T>::GetData()
    {
        return Data;
    }


    //operators block
    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator = (const MDMatrix_gpu<T>& copy)
    {
        Resize(copy.Rows, copy.Cols);
        hipMemcpy(Data, copy.Data, Rows * Cols * sizeof(T), hipMemcpyDeviceToDevice);
        return *this;
    }

    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator = (const MDMatrix_gpu<T>&& other)
    {
        Resize(other.Rows, other.Cols);
        Data = other.Data;
        return *this;
    }

    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator = (T value)
    {
        unsigned int N = Rows * Cols;
        unsigned int threadsPerBlock = _DeviceProps[0].maxThreadsDim[0];
        unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        T* p = Data;

        _FillMatrixValue << <blocksPerGrid, threadsPerBlock >> > (Data, value, Rows, Cols);

        return *this;
    }

    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator = (const T* data)
    {
        hipMemcpy(Data, data, Rows * Cols * sizeof(T), hipMemcpyDeviceToDevice);
        return *this;
    }

    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator = (const void* data)
    {
        hipMemcpy(Data, data, Rows * Cols * sizeof(T), hipMemcpyDeviceToDevice);
        return *this;
    }

    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator += (const MDMatrix_gpu<T>& M)
    {
        if (Rows != M.Rows || Cols != M.Cols) {
            return *this;
        }

        unsigned int N = Rows * Cols;
        unsigned int threadsPerBlock = _DeviceProps[0].maxThreadsDim[0];
        unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        _AddMatrix << <blocksPerGrid, threadsPerBlock >> > (Data, M.Data, Rows * Cols);

        return *this;
    }

    template<class T>
    MDMatrix_gpu<T>& MDMatrix_gpu<T>::operator -= (const MDMatrix_gpu<T>& M)
    {
        if (Rows != M.Rows || Cols != M.Cols) {
            return *this;
        }

        unsigned int N = Rows * Cols;
        unsigned int threadsPerBlock = _DeviceProps[0].maxThreadsDim[0];
        unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        _SubMatrix << <blocksPerGrid, threadsPerBlock >> > (Data, M.Data, Rows * Cols);

        return *this;
    }

    template<class T>
    MDMatrix_gpu<T> MDMatrix_gpu<T>::operator + (const MDMatrix_gpu<T>& M2)
    {
        MDMatrix_gpu<T> res(*this);
        res += M2;
        return res;
    }

    template<class T>
    MDMatrix_gpu<T> MDMatrix_gpu<T>::operator - (const MDMatrix_gpu<T>& M2)
    {
        MDMatrix_gpu<T> res(*this);
        res -= M2;
        return res;
    }

    template<class T>
    MDMatrix_gpu<T> MDMatrix_gpu<T>::operator * (const MDMatrix_gpu<T>& M2)
    {
        _DeviceMatrix<T> A(*this);
        _DeviceMatrix<T> B(M2);
        _DeviceMatrix<T> C(A.Rows, B.Cols);
        dim3 threadsPerBlock(32, 32);
        dim3 blocksPerGrid((A.Cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (B.Rows + threadsPerBlock.y - 1) / threadsPerBlock.y);
        _BadMatMul << <blocksPerGrid, threadsPerBlock >> > (A, B, C);

        MDMatrix_gpu<T> out(C);


        return out;
    }

    template<class T>
    T& MDMatrix_gpu<T>::operator () (int i, int j)
    {
        return Data[i * Cols + j];
    }

    // Общие методы доступа к параметрам матрицы
    // -----------------------------------
    /// Возвращает размерность матрицы
    template<class T>
    int MDMatrix_gpu<T>::GetDimensions() const {
        return 2;
    }

    template<class T>
    int MDMatrix_gpu<T>::GetSize(int i) const
    {
        return (i == 0) ? Rows : ((i == 1) ? Cols : 0);
    }

    template<class T>
    int MDMatrix_gpu<T>::GetSize() const
    {
        return Rows * Cols;
    }

    template<class T>
    bool MDMatrix_gpu<T>::IsEmpty() const
    {
        return (!Rows || !Cols);
    }

    template<class T>
    RDK::MMatrixSize MDMatrix_gpu<T>::GetMatrixSize() const
    {
        MMatrixSize size(Rows, Cols);
        return size;
    }

    template<class T>
    bool MDMatrix_gpu<T>::Resize(const RDK::MMatrixSize& size)
    {
        if (size.GetDimensions() != 2)
            return false;

        Resize(size[0], size[1]);
        return true;
    }

    template<class T>
    int MDMatrix_gpu<T>::GetByteSize() const
    {
        return Rows * Cols * sizeof(T);
    }

    template<class T>
    int MDMatrix_gpu<T>::GetElementByteSize() const
    {
        return sizeof(T);
    }

    template<class T>
    const void* MDMatrix_gpu<T>::GetVoid() const
    {
        return Void;
    }

    template<class T>
    void* MDMatrix_gpu<T>::GetVoid()
    {
        return Void;
    }


    //support block
    //GetDeviceProps()
    template<class T>
    void MDMatrix_gpu<T>::GetDeviceProps()
    {
        hipGetDeviceCount(&_DeviceCount);
        _DeviceProps.resize(_DeviceCount);

        for (int a = 0; a < _DeviceCount; ++a) {
            hipGetDeviceProperties(&_DeviceProps[a], a);
        }
    }

    //print()
    template<class T>
    void MDMatrix_gpu<T>::print() const
    {
        T* temp = new T[Rows * Cols];
        hipMemcpy(temp, Data, Rows * Cols * sizeof(T), hipMemcpyDeviceToHost);
        for (int i = 0; i < Rows; ++i) {
            for (int j = 0; j < Cols; ++j) {
                std::cout << temp[i * Cols + j] << '\t';
            }
            std::cout << '\n';
        }
        std::cout << '\n';
        delete[]temp;
    }

    //
    /*
    template<class T>
    void MDMatrix_gpu<T>::CopyFromCPU(const MMatrixBase* other_I)
    {
        RDK::MDMatrix* other = dynamic_cast<MDMatrix*>(other_I);
        //Resize(other->)
    }*/


}
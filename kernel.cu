﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstdio>

#include "..\..\Core\MDMatrix.h"
#include "..\..\Core\TIM\Timer.h"
#include "MDMatrix_gpu.cu"

using namespace RDK;


int main() {
	setlocale(LC_ALL, "Russian");
	
	std::cout << "CPU operator+\n";
	{
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		std::chrono::duration<double> duration(0);

		
		RDK::MDMatrix<int> a(1000, 1000, 5);
		RDK::MDMatrix<int> b(1000, 1000, 6);

		start = std::chrono::high_resolution_clock::now();
		RDK::MDMatrix<int> c = a+b;
		end = std::chrono::high_resolution_clock::now();

		duration = end - start;
		std::cout << duration.count() << "s\n" << std::endl;
	}

	std::cout << "CPU operator+ in loop\n";
	{
		std::chrono::duration<double> duration(0);
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		for (int i = 0; i < 100; ++i) {

			RDK::MDMatrix<int> a(1000, 1000, 5);
			RDK::MDMatrix<int> b(1000, 1000, 6);

			start = std::chrono::high_resolution_clock::now();
			RDK::MDMatrix<int> c = a + b;
			end = std::chrono::high_resolution_clock::now();

			duration += (end - start);
		}

		duration /= 100;
		std::cout << duration.count() << "s\n" << std::endl;
	}
	

	std::cout << "GPU operator+\n";
	{
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		std::chrono::duration<double> duration(0);

		MDMatrix_gpu<int> a(1000, 1000, 5);
		MDMatrix_gpu<int> b(1000, 1000, 6);

		start = std::chrono::high_resolution_clock::now();
		MDMatrix_gpu<int> c = a + b;
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		duration = end - start;

		std::cout << duration.count() << "s\n" << std::endl;
	}

	std::cout << "GPU operator + in loop\n";
	{
		std::chrono::duration<double> duration(0);
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		for (int i = 0; i < 100; ++i) {

			MDMatrix_gpu<int> a(1000, 1000, 5);
			MDMatrix_gpu<int> b(1000, 1000, 6);

			start = std::chrono::high_resolution_clock::now();

			MDMatrix_gpu<int> c = a + b;

			hipDeviceSynchronize();

			end = std::chrono::high_resolution_clock::now();

			duration += end - start;
		}

		duration /= 100;
		std::cout << duration.count() << "s\n" << std::endl;
	}
	

	std::cout << "CPU operator*\n";
	{
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		std::chrono::duration<double> duration(0);


		RDK::MDMatrix<int> a(1000, 1000, 5);
		RDK::MDMatrix<int> b(1000, 1000, 6);

		start = std::chrono::high_resolution_clock::now();
		RDK::MDMatrix<int> c = a * b;
		end = std::chrono::high_resolution_clock::now();

		duration = end - start;
		std::cout << duration.count() << "s\n" << std::endl;
	}

	/*std::cout << "CPU operator* in loop\n";
	{
		std::chrono::duration<double> duration(0);
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		for (int i = 0; i < 10; ++i) {

			RDK::MDMatrix<int> a(1000, 1000, 5);
			RDK::MDMatrix<int> b(1000, 1000, 6);

			start = std::chrono::high_resolution_clock::now();
			RDK::MDMatrix<int> c = a * b;
			end = std::chrono::high_resolution_clock::now();

			duration += (end - start);
		}

		duration /= 10;
		std::cout << duration.count() << "s\n" << std::endl;
	}*/
	

	std::cout << "GPU operator*\n";
	{
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		std::chrono::duration<double> duration(0);

		MDMatrix_gpu<int> a(1000, 1000, 5);
		MDMatrix_gpu<int> b(1000, 1000, 6);

		start = std::chrono::high_resolution_clock::now();
		MDMatrix_gpu<int> c = a * b;
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		duration = end - start;

		std::cout << duration.count() << "s\n" << std::endl;
	}

	std::cout << "GPU operator * in loop\n";
	{
		std::chrono::duration<double> duration(0);
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		for (int i = 0; i < 100; ++i) {

			MDMatrix_gpu<int> a(1000, 1000, 5);
			MDMatrix_gpu<int> b(1000, 1000, 6);

			start = std::chrono::high_resolution_clock::now();

			MDMatrix_gpu<int> c = a * b;

			hipDeviceSynchronize();

			end = std::chrono::high_resolution_clock::now();

			duration += end - start;
		}

		duration /= 100;
		std::cout << duration.count() << "s\n" << std::endl;
	}


}




